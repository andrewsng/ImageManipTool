#include "hip/hip_runtime.h"
#include "imagemanip.h"
#include "imagemanipcuda.h"
#include "mat2.h"
#include "vec2.h"

#include <iostream>
#define _USE_MATH_DEFINES
#include <cmath>
#include <cstddef>
using std::size_t;
#include <algorithm>
using std::max;
using std::min;
using std::swap;
#include <vector>
using std::vector;
using FilterType1D = vector<float>;
using FilterType2D = vector<vector<float>>;


__global__ void transformKernel(float * input, float * output, float minX, float minY, 
                                float dx, float dy, int sWidth, int sHeight, int dWidth, int dHeight, float * mat)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    float tx = mat[0] * (minX + x * dx) + mat[1] * (minY + y * dy);
    float ty = mat[2] * (minX + x * dx) + mat[3] * (minY + y * dy);

    if (0.0f <= tx && tx < float(sWidth) - 1.0f &&
        0.0f <= ty && ty < float(sHeight) - 1.0f)
    {
        output[x*3+0 + ((dHeight-1-y)*3 * dWidth)] = input[int(tx)*3+0 + ((sHeight-1-int(ty)) * 3 * sWidth)];
        output[x*3+1 + ((dHeight-1-y)*3 * dWidth)] = input[int(tx)*3+1 + ((sHeight-1-int(ty)) * 3 * sWidth)];
        output[x*3+2 + ((dHeight-1-y)*3 * dWidth)] = input[int(tx)*3+2 + ((sHeight-1-int(ty)) * 3 * sWidth)];
    }
}


// Image::Pixel bilinearInterp(const Image & image, float x, float y)
// {
//     int x0 = min(int(x), image.width() - 1);
//     int y0 = min(int(y), image.height() - 1);
//     int x1 = min(x0 + 1, image.width() - 1);
//     int y1 = min(y0 + 1, image.height() - 1);
//     Image::Pixel c00 = image.getColor(x0, y0);
//     Image::Pixel c01 = image.getColor(x0, y1);
//     Image::Pixel c10 = image.getColor(x1, y0);
//     Image::Pixel c11 = image.getColor(x1, y1);
//     float biasX = x - x0;
//     float biasY = y - y0;

//     Image::Pixel result;
//     result.r = (1.0f - biasY) * ((1.0f - biasX) * c00.r + biasX * c10.r) + biasY * ((1.0f - biasX) * c01.r + biasX * c11.r);
//     result.g = (1.0f - biasY) * ((1.0f - biasX) * c00.g + biasX * c10.g) + biasY * ((1.0f - biasX) * c01.g + biasX * c11.g);
//     result.b = (1.0f - biasY) * ((1.0f - biasX) * c00.b + biasX * c10.b) + biasY * ((1.0f - biasX) * c01.b + biasX * c11.b);

//     return result;
// }


Image transformCuda(const Image & image, Mat2 transform)
{
    float maxX = float(image.width() - 1);
    float maxY = float(image.height() - 1);
    Vec2 topLeft(0.0f, maxY);
    Vec2 topRight(maxX, maxY);
    Vec2 botLeft(0.0f, 0.0f);
    Vec2 botRight(maxX, 0.0f);

    topLeft = transform * topLeft;
    topRight = transform * topRight;
    botLeft = transform * botLeft;
    botRight = transform * botRight;

    float newMaxX = max({ topLeft.x, topRight.x, botLeft.x, botRight.x });
    float newMinX = min({ topLeft.x, topRight.x, botLeft.x, botRight.x });
    float newMaxY = max({ topLeft.y, topRight.y, botLeft.y, botRight.y });
    float newMinY = min({ topLeft.y, topRight.y, botLeft.y, botRight.y });

    Image transformed(newMaxX - newMinX + 1, newMaxY - newMinY + 1);

    float dx = (newMaxX - newMinX) / transformed.width();
    float dy = (newMaxY - newMinY) / transformed.height();

    Image source;
    // float sizeRatio = float(transformed.width() * transformed.height()) / (image.width() * image.height());
    // if (sizeRatio < 0.25)
    //     source = gaussianBlurSeparableCuda(image, 9, 3);
    // else if (sizeRatio > 1.5f)
    //     source = gaussianBlurSeparableCuda(image, 3, 1);
    // else
        source = image;

    Mat2 transformInv = transform.inverse();

    float * input;
    hipMalloc(&input, source.width() * source.height() * sizeof(float) * 3);
    hipMemcpy(input, &source(0, 0), source.width() * source.height() * sizeof(float) * 3, hipMemcpyHostToDevice);
    float * output;
    hipMalloc(&output, transformed.width() * transformed.height() * sizeof(float) * 3);
    float * mat;
    hipMalloc(&mat, 4 * sizeof(float));
    hipMemcpy(mat, &transformInv[0][0], 4 * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 gridSize((transformed.width()  + blockSize.x - 1) / blockSize.x,
                  (transformed.height() + blockSize.y - 1) / blockSize.y);

    transformKernel<<<gridSize, blockSize>>>(input, output, newMinX, newMinY, dx, dy,
        source.width(), source.height(), transformed.width(), transformed.height(), mat);

    hipDeviceSynchronize();
    hipMemcpy(&transformed(0, 0), output, transformed.width() * transformed.height() * sizeof(float) * 3, hipMemcpyDeviceToHost);

    hipFree(input);
    hipFree(output);
    hipFree(mat);

    return transformed;
}


Image rotateCuda(const Image & image, float angleDeg)
{
    float angleRad = M_PI * angleDeg / 180.0f;
    float cosAngle = cosf(angleRad);
    float sinAngle = sinf(angleRad);
    Mat2 rotMat = {
        { cosAngle, -sinAngle },
        { sinAngle,  cosAngle }
    };

    return transformCuda(image, rotMat);   
}


Image scaleCuda(const Image & image, float scaleX, float scaleY)
{
    Mat2 scaleMat = {
        { scaleX, 0.0f   },
        { 0.0f,   scaleY }
    };

    return transformCuda(image, scaleMat);
}


__device__ int clamp(int x, int a, int b)
{
    if (x < a)
        return a;
    if (x > b)
        return b;
    return x;
}


__global__ void convolve2DKernel(float * input, float * output, int width, int height, float * filter, int size)
{
    int r = (size - 1) / 2;
    float sumR = 0.0f;
    float sumG = 0.0f;
    float sumB = 0.0f;
    
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    for (int j = y-r; j <= y+r; ++j)
    {
        for (int i = x-r; i <= x+r; ++i)
        {
            float weight = filter[(j-y+r) * size + (i-x+r)];
            int ci = clamp(i, 0, width-1);
            int cj = clamp(j, 0, height-1);

            sumR += input[cj * width*3 + ci*3 + 0] * weight;
            sumG += input[cj * width*3 + ci*3 + 1] * weight;
            sumB += input[cj * width*3 + ci*3 + 2] * weight;
        }
    }

    output[y * width*3 + x*3 + 0] = sumR;
    output[y * width*3 + x*3 + 1] = sumG;
    output[y * width*3 + x*3 + 2] = sumB;
}


__global__ void printFilter(float * filter, int size)
{
    int x = 0;
    int y = 0;
    int r = (size - 1) / 2;
    for (int j = y-r; j <= y+r; ++j)
    {
        for (int i = x-r; i <= x+r; ++i)
        {
            float weight = filter[(i-x+r) * size + (j-y+r)];
            printf("%f ", weight);
        }
        printf("\n");
    }
}


Image convolveImage2DCuda(const Image & image, const FilterType1D & filter)
{
    Image convolved(image.width(), image.height());

    int size = sqrt(filter.size());

    float * input;
    hipMalloc(&input, image.width() * image.height() * sizeof(float) * 3);
    hipMemcpy(input, &image(0, 0), image.width() * image.height() * sizeof(float) * 3, hipMemcpyHostToDevice);
    float * output;
    hipMalloc(&output, convolved.width() * convolved.height() * sizeof(float) * 3);
    float * filter2D;
    hipMalloc(&filter2D, size * size * sizeof(float));
    hipMemcpy(filter2D, &filter[0], size * size * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 gridSize((convolved.width()  + blockSize.x - 1) / blockSize.x,
                  (convolved.height() + blockSize.y - 1) / blockSize.y);

    convolve2DKernel<<<gridSize, blockSize>>>(input, output, image.width(), image.height(), filter2D, size);

    hipDeviceSynchronize();
    hipMemcpy(&convolved(0, 0), output, convolved.width() * convolved.height() * sizeof(float) * 3, hipMemcpyDeviceToHost);

    hipFree(input);
    hipFree(output);
    hipFree(filter2D);

    return convolved;
}


__global__ void convolve1DKernel(float * input, float * output, int width, int height, float * filter, int size, bool vert)
{
    int r = (size - 1) / 2;
    float sumR = 0.0f;
    float sumG = 0.0f;
    float sumB = 0.0f;
    
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (vert)
    {
        int tmp = x;
        x = y;
        y = tmp;
    }
    for (int i = x-r; i <= x+r; ++i)
    {
        float weight = filter[(i-x+r)];
        int ci = clamp(i, 0, width-1);
        if (vert)
        {
            ci = clamp(i, 0, height-1);
            sumR += input[ci * width*3 + y*3 + 0] * weight;
            sumG += input[ci * width*3 + y*3 + 1] * weight;
            sumB += input[ci * width*3 + y*3 + 2] * weight;
        }
        else
        {
            ci = clamp(i, 0, width-1);
            sumR += input[y * width*3 + ci*3 + 0] * weight;
            sumG += input[y * width*3 + ci*3 + 1] * weight;
            sumB += input[y * width*3 + ci*3 + 2] * weight;
        }
    }

    if (vert)
    {
        output[x * width*3 + y*3 + 0] = sumR;
        output[x * width*3 + y*3 + 1] = sumG;
        output[x * width*3 + y*3 + 2] = sumB;
    }
    else
    {
        output[y * width*3 + x*3 + 0] = sumR;
        output[y * width*3 + x*3 + 1] = sumG;
        output[y * width*3 + x*3 + 2] = sumB;
    }
}


Image convolveImage1DCuda(const Image & image, const FilterType1D & filter)
{
    Image middle(image.width(), image.height());

    int size = filter.size();
    
    float * input;
    hipMalloc(&input, image.width() * image.height() * sizeof(float) * 3);
    hipMemcpy(input, &image(0, 0), image.width() * image.height() * sizeof(float) * 3, hipMemcpyHostToDevice);

    float * output;
    hipMalloc(&output, middle.width() * middle.height() * sizeof(float) * 3);
    
    float * filter1D;
    hipMalloc(&filter1D, size * sizeof(float));
    hipMemcpy(filter1D, &filter[0], size * sizeof(float), hipMemcpyHostToDevice);
    
    dim3 blockSize(16, 16);
    dim3 gridSize((middle.width()  + blockSize.x - 1) / blockSize.x,
                  (middle.height() + blockSize.y - 1) / blockSize.y);

    convolve1DKernel<<<gridSize, blockSize>>>(input, output, image.width(), image.height(), filter1D, size, false);

    hipDeviceSynchronize();
    
    //hipMemcpy(input, output, middle.width() * middle.height() * sizeof(float) * 3, hipMemcpyDeviceToDevice);

    convolve1DKernel<<<gridSize, blockSize>>>(output, input, middle.width(), middle.height(), filter1D, size, true);
    
    hipDeviceSynchronize();
    
    Image convolved(middle.width(), middle.height());
    hipMemcpy(&convolved(0, 0), input, convolved.width() * convolved.height() * sizeof(float) * 3, hipMemcpyDeviceToHost);

    hipFree(input);
    hipFree(output);
    hipFree(filter1D);

    return convolved;
}


Image boxBlurCuda(const Image & image, int radius)
{
    int size = 2 * radius + 1;
    float weight = 1.0f / (size * size);
    FilterType1D boxFilter(size*size, weight);

    return convolveImage2DCuda(image, boxFilter);
}


Image boxBlurSeparableCuda(const Image & image, int radius)
{
    int size = 2 * radius + 1;
    float weight = 1.0f / size;
    FilterType1D boxFilter(size, weight);

    return convolveImage1DCuda(image, boxFilter);
}


Image gaussianBlurCuda(const Image & image, int radius, float stddev)
{
    int size = 2 * radius + 1;
    float s = 2 * stddev * stddev;
    FilterType1D gaussianFilter(size*size, 0.0f);
    for (int y = 0; y < size; ++y)
    {
        for (int x = 0; x < size; ++x)
        {
            float x2 = x - radius;
            float y2 = y - radius;
            float G = expf(-(x2*x2 + y2*y2) / s) / (M_PI * s);
            gaussianFilter[y*size+x] = G;
        }
    }

    return convolveImage2DCuda(image, gaussianFilter);
}


Image gaussianBlurSeparableCuda(const Image & image, int radius, float stddev)
{
    int size = 2 * radius + 1;
    float s = 2 * stddev * stddev;
    FilterType1D gaussianFilter(size, 0.0f);
    for (int x = 0; x < size; ++x)
    {
        float x2 = x - radius;
        float G = expf(-(x2 * x2) / s) / sqrtf(M_PI * s);
        gaussianFilter[x] = G;
    }

    return convolveImage1DCuda(image, gaussianFilter);
}

