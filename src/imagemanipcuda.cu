#include "hip/hip_runtime.h"
// imagemanipcuda.cu
// Andrew S. Ng
// Started: 2020-12-01
// Updated: 2020-12-07
//
// For CS 301 Fall 2020
// Source for image processing operations in CUDA

#include "imagemanip.h"
#include "imagemanipcuda.h"
#include "mat2.h"
#include "vec2.h"

#include <iostream>
#define _USE_MATH_DEFINES
#include <cmath>
#include <cstddef>
using std::size_t;
#include <algorithm>
using std::max;
using std::min;
using std::swap;
#include <vector>
using std::vector;
using FilterType1D = vector<float>;
using FilterType2D = vector<vector<float>>;


__device__ void bilinearInterp(float * input, float x, float y, int width, int height, float * color)
{
    int x0 = int(x) < width-1 ? int(x) : width-1;
    int y0 = int(y) < height-1 ? int(y) : height-1;
    int x1 = x0+1 < width-1 ? x0+1 : width-1;
    int y1 = y0+1 < height-1 ? y0+1 : height-1;
    float * c00 = &input[y0 * width*3 + x0*3];
    float * c01 = &input[y1 * width*3 + x0*3];
    float * c10 = &input[y0 * width*3 + x1*3];
    float * c11 = &input[y1 * width*3 + x1*3];
    float biasX = x - x0;
    float biasY = y - y0;

    color[0] = (1.0f - biasY) * ((1.0f - biasX) * c00[0] + biasX * c10[0]) + biasY * ((1.0f - biasX) * c01[0] + biasX * c11[0]);
    color[1] = (1.0f - biasY) * ((1.0f - biasX) * c00[1] + biasX * c10[1]) + biasY * ((1.0f - biasX) * c01[1] + biasX * c11[1]);
    color[2] = (1.0f - biasY) * ((1.0f - biasX) * c00[2] + biasX * c10[2]) + biasY * ((1.0f - biasX) * c01[2] + biasX * c11[2]);
}


__global__ void transformKernel(float * input, float * output, float minX, float minY, 
                                float dx, float dy, int iWidth, int iHeight, int oWidth, int oHeight, float * mat)
{                            
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < 0 || x > oWidth-1 || y < 0 || y > oHeight-1)
        return;

    float tx = mat[0] * (minX + x * dx) + mat[1] * (minY + y * dy);    
    float ty = mat[2] * (minX + x * dx) + mat[3] * (minY + y * dy);

    if (-0.5f <= tx && tx <= float(iWidth) - 0.5f &&
        -0.5f <= ty && ty <= float(iHeight) - 0.5f)
    {
        int out = (oHeight-1-y) * oWidth*3 + x*3;
        // int in = (iHeight-1-int(ty)) * iWidth*3 + int(tx)*3;
        // output[out+0] = input[in+0];
        // output[out+1] = input[in+1];
        // output[out+2] = input[in+2];

        float color[3];
        bilinearInterp(input, tx+0.5f, iHeight-1-ty+0.5f, iWidth, iHeight, color);
        output[out+0] = color[0];
        output[out+1] = color[1];
        output[out+2] = color[2];
    }    
}    


Image transformCuda(const Image & image, Mat2 transform)
{
    float maxX = float(image.width() - 1);
    float maxY = float(image.height() - 1);
    Vec2 topLeft(0.0f, maxY);
    Vec2 topRight(maxX, maxY);
    Vec2 botLeft(0.0f, 0.0f);
    Vec2 botRight(maxX, 0.0f);

    topLeft = transform * topLeft;
    topRight = transform * topRight;
    botLeft = transform * botLeft;
    botRight = transform * botRight;

    float newMaxX = max({ topLeft.x, topRight.x, botLeft.x, botRight.x });
    float newMinX = min({ topLeft.x, topRight.x, botLeft.x, botRight.x });
    float newMaxY = max({ topLeft.y, topRight.y, botLeft.y, botRight.y });
    float newMinY = min({ topLeft.y, topRight.y, botLeft.y, botRight.y });

    Image transformed(newMaxX - newMinX + 1, newMaxY - newMinY + 1);

    float dx = (newMaxX - newMinX) / transformed.width();
    float dy = (newMaxY - newMinY) / transformed.height();

    Image source;
    float sizeRatio = float(transformed.width() * transformed.height()) / (image.width() * image.height());
    if (sizeRatio < 0.25)
        source = gaussianBlurSeparableCuda(image, 9, 3);
    // else if (sizeRatio > 1.5f)
    //     source = gaussianBlurSeparableCuda(image, 3, 1);
    else
        source = image;

    Mat2 transformInv = transform.inverse();

    float * input;
    hipMalloc(&input, source.width() * source.height() * sizeof(float) * 3);
    hipMemcpy(input, &source(0, 0), source.width() * source.height() * sizeof(float) * 3, hipMemcpyHostToDevice);
    float * output;
    hipMalloc(&output, transformed.width() * transformed.height() * sizeof(float) * 3);
    float * mat;
    hipMalloc(&mat, 4 * sizeof(float));
    hipMemcpy(mat, &transformInv[0][0], 4 * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 gridSize((transformed.width()  + blockSize.x - 1) / blockSize.x,
                  (transformed.height() + blockSize.y - 1) / blockSize.y);

    transformKernel<<<gridSize, blockSize>>>(input, output, newMinX, newMinY, dx, dy,
        source.width(), source.height(), transformed.width(), transformed.height(), mat);

    hipDeviceSynchronize();
    hipMemcpy(&transformed(0, 0), output, transformed.width() * transformed.height() * sizeof(float) * 3, hipMemcpyDeviceToHost);

    hipFree(input);
    hipFree(output);
    hipFree(mat);

    return transformed;
}


Image rotateCuda(const Image & image, float angleDeg)
{
    float angleRad = M_PI * angleDeg / 180.0f;
    float cosAngle = cosf(angleRad);
    float sinAngle = sinf(angleRad);
    Mat2 rotMat = {
        { cosAngle, -sinAngle },
        { sinAngle,  cosAngle }
    };

    return transformCuda(image, rotMat);   
}


Image scaleCuda(const Image & image, float scaleX, float scaleY)
{
    Mat2 scaleMat = {
        { scaleX, 0.0f   },
        { 0.0f,   scaleY }
    };

    return transformCuda(image, scaleMat);
}


__device__ int clamp(int x, int a, int b)
{
    if (b < a)
    {
        int tmp = a;
        a = b;
        b = tmp;
    }
    if (x < a)
        return a;
    if (x > b)
        return b;
    return x;
}


__global__ void convolve2DKernel(float * input, float * output, int width, int height, float * filter, int size)
{
    int r = (size - 1) / 2;
    float sumR = 0.0f;
    float sumG = 0.0f;
    float sumB = 0.0f;
    
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < 0 || x > width-1 || y < 0 || y > height-1)
        return;

    for (int j = y-r; j <= y+r; ++j)
    {
        for (int i = x-r; i <= x+r; ++i)
        {
            float weight = filter[(j-y+r) * size + (i-x+r)];
            int ci = clamp(i, 0, width-1);
            int cj = clamp(j, 0, height-1);

            sumR += input[cj * width*3 + ci*3 + 0] * weight;
            sumG += input[cj * width*3 + ci*3 + 1] * weight;
            sumB += input[cj * width*3 + ci*3 + 2] * weight;
        }
    }

    output[y * width*3 + x*3 + 0] = sumR;
    output[y * width*3 + x*3 + 1] = sumG;
    output[y * width*3 + x*3 + 2] = sumB;
}


__global__ void printFilter(float * filter, int size)
{
    int x = 0;
    int y = 0;
    int r = (size - 1) / 2;
    for (int j = y-r; j <= y+r; ++j)
    {
        for (int i = x-r; i <= x+r; ++i)
        {
            float weight = filter[(i-x+r) * size + (j-y+r)];
            printf("%f ", weight);
        }
        printf("\n");
    }
    printf("\n");
}


Image convolveImage2DCuda(const Image & image, const FilterType1D & filter)
{
    Image convolved(image.width(), image.height());

    int size = sqrt(filter.size());

    float * input;
    hipMalloc(&input, image.width() * image.height() * sizeof(float) * 3);
    hipMemcpy(input, &image(0, 0), image.width() * image.height() * sizeof(float) * 3, hipMemcpyHostToDevice);
    float * output;
    hipMalloc(&output, convolved.width() * convolved.height() * sizeof(float) * 3);
    float * filter2D;
    hipMalloc(&filter2D, size * size * sizeof(float));
    hipMemcpy(filter2D, &filter[0], size * size * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 gridSize((convolved.width()  + blockSize.x - 1) / blockSize.x,
                  (convolved.height() + blockSize.y - 1) / blockSize.y);

    convolve2DKernel<<<gridSize, blockSize>>>(input, output, image.width(), image.height(), filter2D, size);

    hipDeviceSynchronize();
    hipMemcpy(&convolved(0, 0), output, convolved.width() * convolved.height() * sizeof(float) * 3, hipMemcpyDeviceToHost);

    hipFree(input);
    hipFree(output);
    hipFree(filter2D);

    return convolved;
}


__global__ void convolve1DKernel(float * input, float * output, int width, int height, float * filter, int size, bool vert)
{
    int r = (size - 1) / 2;
    float sumR = 0.0f;
    float sumG = 0.0f;
    float sumB = 0.0f;
    
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < 0 || x > width-1 || y < 0 || y > height-1)
        return;

    if (vert)
    {
        int tmp = x;
        x = y;
        y = tmp;
    }
    for (int i = x-r; i <= x+r; ++i)
    {
        float weight = filter[(i-x+r)];
        int ci = clamp(i, 0, width-1);
        if (vert)
        {
            ci = clamp(i, 0, height-1);
            sumR += input[ci * width*3 + y*3 + 0] * weight;
            sumG += input[ci * width*3 + y*3 + 1] * weight;
            sumB += input[ci * width*3 + y*3 + 2] * weight;
        }
        else
        {
            ci = clamp(i, 0, width-1);
            sumR += input[y * width*3 + ci*3 + 0] * weight;
            sumG += input[y * width*3 + ci*3 + 1] * weight;
            sumB += input[y * width*3 + ci*3 + 2] * weight;
        }
    }

    if (vert)
    {
        output[x * width*3 + y*3 + 0] = sumR;
        output[x * width*3 + y*3 + 1] = sumG;
        output[x * width*3 + y*3 + 2] = sumB;
    }
    else
    {
        output[y * width*3 + x*3 + 0] = sumR;
        output[y * width*3 + x*3 + 1] = sumG;
        output[y * width*3 + x*3 + 2] = sumB;
    }
}


Image convolveImage1DCuda(const Image & image, const FilterType1D & filter)
{
    Image middle(image.width(), image.height());

    int size = filter.size();
    
    float * input;
    hipMalloc(&input, image.width() * image.height() * sizeof(float) * 3);
    hipMemcpy(input, &image(0, 0), image.width() * image.height() * sizeof(float) * 3, hipMemcpyHostToDevice);

    float * output;
    hipMalloc(&output, middle.width() * middle.height() * sizeof(float) * 3);
    
    float * filter1D;
    hipMalloc(&filter1D, size * sizeof(float));
    hipMemcpy(filter1D, &filter[0], size * sizeof(float), hipMemcpyHostToDevice);
    
    dim3 blockSize(16, 16);
    dim3 gridSize((middle.width()  + blockSize.x - 1) / blockSize.x,
                  (middle.height() + blockSize.y - 1) / blockSize.y);

    convolve1DKernel<<<gridSize, blockSize>>>(input, output, image.width(), image.height(), filter1D, size, false);

    hipDeviceSynchronize();

    convolve1DKernel<<<gridSize, blockSize>>>(output, input, middle.width(), middle.height(), filter1D, size, true);
    
    hipDeviceSynchronize();
    
    Image convolved(middle.width(), middle.height());
    hipMemcpy(&convolved(0, 0), input, convolved.width() * convolved.height() * sizeof(float) * 3, hipMemcpyDeviceToHost);

    hipFree(input);
    hipFree(output);
    hipFree(filter1D);

    return convolved;
}


Image boxBlurCuda(const Image & image, int radius)
{
    int size = 2 * radius + 1;
    float weight = 1.0f / (size * size);
    FilterType1D boxFilter(size*size, weight);

    return convolveImage2DCuda(image, boxFilter);
}


Image boxBlurSeparableCuda(const Image & image, int radius)
{
    int size = 2 * radius + 1;
    float weight = 1.0f / size;
    FilterType1D boxFilter(size, weight);

    return convolveImage1DCuda(image, boxFilter);
}


Image gaussianBlurCuda(const Image & image, int radius, float stddev)
{
    int size = 2 * radius + 1;
    float s = 2 * stddev * stddev;
    FilterType1D gaussianFilter(size*size, 0.0f);
    for (int y = 0; y < size; ++y)
    {
        for (int x = 0; x < size; ++x)
        {
            float x2 = x - radius;
            float y2 = y - radius;
            float G = expf(-(x2*x2 + y2*y2) / s) / (M_PI * s);
            gaussianFilter[y*size+x] = G;
        }
    }

    return convolveImage2DCuda(image, gaussianFilter);
}


Image gaussianBlurSeparableCuda(const Image & image, int radius, float stddev)
{
    int size = 2 * radius + 1;
    float s = 2 * stddev * stddev;
    FilterType1D gaussianFilter(size, 0.0f);
    for (int x = 0; x < size; ++x)
    {
        float x2 = x - radius;
        float G = expf(-(x2 * x2) / s) / sqrtf(M_PI * s);
        gaussianFilter[x] = G;
    }

    return convolveImage1DCuda(image, gaussianFilter);
}

